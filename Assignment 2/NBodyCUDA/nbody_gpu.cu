#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "NBody.h"
#include "NBodyVisualiser.h"
#include "nbody_data.h"

#define THREADS_PER_BLOCK 32

// This will output the proper CUDA error strings in the event
// that a CUDA host call returns an error
#define checkCudaError(val) check((val), #val, __LINE__)

// External variables defined in `nbody_args`
extern unsigned int N;
extern unsigned int D;
extern unsigned int I;

// External variables defined in `NBody`
extern unsigned int grid_size;
extern float normalising_factor;
extern size_t nbodies_size;
extern size_t activity_map_size;

static nbody *h_nbodies, *d_nbodies;
static float *h_activity_map, *d_activity_map;

dim3 nbodies_blocksPerGrid;
dim3 activity_map_blocksPerGrid;

// Function declarations
static void step_gpu() noexcept;
static void allocate_memory() noexcept;
static void check(hipError_t err, char const *func, int line) noexcept;

__global__ void parallelise_each_body(nbody *d_nbodies, float *d_activity_map, const unsigned int N, const unsigned int D) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        /* Force */
        float sum_x = 0, sum_y = 0;

        for (unsigned int j = 0; j < N; ++j) {
            const float dist_x = d_nbodies[j].x - d_nbodies[i].x;
            const float dist_y = d_nbodies[j].y - d_nbodies[i].y;
            const float mag_add_soft = dist_x * dist_x + dist_y * dist_y + SOFTENING_SQUARE;
            const float m_div_soft = d_nbodies[j].m / (mag_add_soft * sqrtf(mag_add_soft));

            sum_x += m_div_soft * dist_x;
            sum_y += m_div_soft * dist_y;
        }

        /* Movement */
        // Calculate position vector, do this first as it depends on current velocity
        d_nbodies[i].x += dt * d_nbodies[i].vx;
        d_nbodies[i].y += dt * d_nbodies[i].vy;

        // Calculate velocity vector, force and acceleration are computed together
        d_nbodies[i].vx += dt_MUL_G * sum_x;
        d_nbodies[i].vy += dt_MUL_G * sum_y;

        /* compute the position for a body in the `activity_map`
         * and increase the corresponding body count */
        const unsigned int col = static_cast<unsigned int>(d_nbodies[i].x * static_cast<float>(D));
        const unsigned int row = static_cast<unsigned int>(d_nbodies[i].y * static_cast<float>(D));

        // Do not update `activity_map` if n-body is out of grid area
        if (row < D && col < D) {
            atomicAdd(&d_activity_map[D * row + col], 1);
        }
    }
}

__global__ void normalise_activity_map(float *d_activity_map, const unsigned int grid_size, const float normalising_factor) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < grid_size) {
        d_activity_map[i] *= normalising_factor;
    }
}

int main_gpu() noexcept {
    // Allocate any host memory and device memory
    allocate_memory();

    // Initialise host data
    initialise_data_aos(h_nbodies);

    // Copy host data to device
    checkCudaError(hipMemcpy(d_nbodies, h_nbodies, nbodies_size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_activity_map, h_activity_map, activity_map_size, hipMemcpyHostToDevice));

    // Calculate the required blocks
    nbodies_blocksPerGrid = { (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1 };
    activity_map_blocksPerGrid = { (grid_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1 };

    if (I == 0) {
        initViewer(N, D, CUDA, &step_gpu);
        setNBodyPositions(d_nbodies);
        setHistogramData(d_activity_map);
        startVisualisationLoop();
    } else {
        // CUDA event creation
        hipEvent_t start, stop;
        checkCudaError(hipEventCreate(&start));
        checkCudaError(hipEventCreate(&stop));

        // Record timing of step_gpu()
        checkCudaError(hipEventRecord(start));
        for (unsigned int i = 0; i < I; ++i) {
            step_gpu();
        }
        checkCudaError(hipEventRecord(stop));
        checkCudaError(hipEventSynchronize(stop));

        // Output total time taken
        float ms;
        checkCudaError(hipEventElapsedTime(&ms, start, stop));

        const int seconds = static_cast<int>(ms) / 1000;
        printf("Execution time was %d seconds %d milliseconds\n", seconds, static_cast<int>(ms) % 1000);
    }

    // Free host memory
    free(h_nbodies);
    free(h_activity_map);

    // Free device memory
    checkCudaError(hipDeviceReset());

    return 0;
}

/**
 * Perform the main simulation of the NBody system on the CPU
 */
static void step_gpu() noexcept {
    // Clear the activity map of previous step
    checkCudaError(hipMemset(d_activity_map, 0, activity_map_size));

    parallelise_each_body << <nbodies_blocksPerGrid, THREADS_PER_BLOCK >> > (d_nbodies, d_activity_map, N, D);
    normalise_activity_map << <activity_map_blocksPerGrid, THREADS_PER_BLOCK >> > (d_activity_map, grid_size, normalising_factor);
}

/**
 * Allocate required memory for host and device
 */
static void allocate_memory() noexcept {
    // Host memory
    h_nbodies = static_cast<nbody *>(malloc(nbodies_size));
    if (h_nbodies == nullptr) {
        fprintf(stderr, "error: failed to allocate memory: h_nbodies\n");
        exit(EXIT_FAILURE);
    }

    h_activity_map = static_cast<float *>(malloc(activity_map_size));
    if (h_activity_map == nullptr) {
        fprintf(stderr, "error: failed to allocate memory: h_activity_map");
        exit(EXIT_FAILURE);
    }

    // Device memory
    checkCudaError(hipMalloc((void **)&d_nbodies, nbodies_size));
    checkCudaError(hipMalloc((void **)&d_activity_map, activity_map_size));
}

/**
 * --- Used by checkCudaError macro ---
 * Check for CUDA error. Exit with failure if an error occurred.
 *
 * @param err The hipError_t value
 * @param func The line of code that generated the error
 * @param line The line number
 */
static void check(hipError_t err, char const *func, int line) noexcept {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error \"%s\" at line %d: %s\n", hipGetErrorString(err), line, func);
        exit(EXIT_FAILURE);
    }
}
