#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <omp.h>

#include "NBody.h"
#include "NBodyVisualiser.h"
#include "nbody_data.h"

// External variables defined in `nbody_args`
extern unsigned int N;
extern unsigned int D;
extern MODE M;
extern unsigned int I;

// External variables defined in `NBody`
extern unsigned int grid_size;
extern float normalising_factor;
extern size_t nbodies_size;
extern size_t activity_map_size;

static nbody *nbodies;
static float *activity_map;

// Function declarations
static void step_cpu() noexcept;
static void allocate_memory() noexcept;

/**
 * Entry point of program for CPU and OPENMP mode
 *
 * @return int The program exit code
 */
int main_cpu() {
    // Allocate any heap memory
    allocate_memory();

    // Initialise N-bodies data
    initialise_data_aos(nbodies);

    if (I == 0) {
        // Start the visualiser
        initViewer(N, D, M, &step_cpu);
        setNBodyPositions(nbodies);
        setHistogramData(activity_map);
        startVisualisationLoop();
    } else {
        // Perform a fixed number of simulation steps, then output the timing results
        const double start = omp_get_wtime();

        for (unsigned i = 0; i < I; ++i) {
            step_cpu();
        }

        const double seconds = omp_get_wtime() - start;
        const int milliseconds = static_cast<int>((seconds - static_cast<int>(seconds)) * 1000);

        printf("Execution time %d seconds %d milliseconds\n", static_cast<int>(seconds), milliseconds);
    }

    // Free memory
    free(nbodies);
    free(activity_map);

    return 0;
}

/**
 * Perform the main simulation of the NBody system on the CPU
 */
static void step_cpu() noexcept {
    int i;

    // Clear the activity map of previous step
    memset(activity_map, 0, activity_map_size);

#pragma omp parallel for schedule(static) default(none) shared(N, D, nbodies, activity_map) if (M == OPENMP)
    for (i = 0; i < static_cast<int>(N); ++i) {
        /* Force */
        float sum_x = 0, sum_y = 0;

        for (unsigned int j = 0; j < N; ++j) {
            const float dist_x = nbodies[j].x - nbodies[i].x;
            const float dist_y = nbodies[j].y - nbodies[i].y;
            const float mag_add_soft = dist_x * dist_x + dist_y * dist_y + SOFTENING_SQUARE;
            const float m_div_soft = nbodies[j].m / (mag_add_soft * sqrtf(mag_add_soft));

            sum_x += m_div_soft * dist_x;
            sum_y += m_div_soft * dist_y;
        }

        /* Movement */
        // Calculate position vector, do this first as it depends on current velocity
        nbodies[i].x += dt * nbodies[i].vx;
        nbodies[i].y += dt * nbodies[i].vy;

        // Calculate velocity vector, force and acceleration are computed together
        nbodies[i].vx += dt_MUL_G * sum_x;
        nbodies[i].vy += dt_MUL_G * sum_y;

        /* compute the position for a body in the `activity_map`
         * and increase the corresponding body count */
        const unsigned int col = static_cast<unsigned int>(nbodies[i].x * static_cast<float>(D));
        const unsigned int row = static_cast<unsigned int>(nbodies[i].y * static_cast<float>(D));
        const unsigned int cell = static_cast<unsigned int>(D * row + col);

        // Do not update `activity_map` if n-body is out of grid area
        if (cell < grid_size) {
            if (M == OPENMP) {
#pragma omp atomic
                ++activity_map[cell];
            } else {
                ++activity_map[cell];
            }
        }
    }

    /* Loop through the `activity_map` to normalise the body counts */
    for (i = 0; i < static_cast<int>(grid_size); ++i) {
        activity_map[i] *= normalising_factor;
    }
}

/**
 * Allocate required memory
 */
static void allocate_memory() noexcept {
    nbodies = static_cast<nbody *>(malloc(nbodies_size));
    if (nbodies == nullptr) {
        fprintf(stderr, "error: failed to allocate memory: nbodies\n");
        exit(EXIT_FAILURE);
    }

    activity_map = static_cast<float *>(malloc(activity_map_size));
    if (activity_map == nullptr) {
        fprintf(stderr, "error: failed to allocate memory: activity_map");
        exit(EXIT_FAILURE);
    }
}
