#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cctype>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

typedef enum {
    CALCULATOR_ADD,
    CALCULATOR_SUB,
    CALCULATOR_DIV,
    CALCULATOR_MUL
} CALCULATOR_COMMANDS;

typedef enum {
    INPUT_RANDOM,
    INPUT_LINEAR
} INPUT_TYPE;

#define SAMPLES 262144
#define TPB 256
#define NUM_STREAMS 2
#define FILE_BUFFER_SIZE 32
#define MAX_COMMANDS 32
#define INPUT INPUT_LINEAR

__constant__ CALCULATOR_COMMANDS d_commands[MAX_COMMANDS];
__constant__ float d_operands[MAX_COMMANDS];

int readCommandsFromFile(CALCULATOR_COMMANDS *commands, float *operands);
void initInput(float *input);
void checkCUDAError(const char *msg);
int readLine(FILE *f, char buffer[]);
void cudaCalculatorDefaultStream(CALCULATOR_COMMANDS *commands, float *operands, int num_commands);
void cudaCalculatorNStream1(CALCULATOR_COMMANDS *commands, float *operands, int num_commands);
void cudaCalculatorNStream2(CALCULATOR_COMMANDS *commands, float *operands, int num_commands);
int checkResults(const float *h_input, float *h_output, const CALCULATOR_COMMANDS *commands, const float *operands, int num_commands);

__global__ void parallelCalculator(float *input, float *output, int num_commands) {
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;

    //get input
    float out = input[idx];

    //apply commands
    for (int i = 0; i < num_commands; i++) {
        const CALCULATOR_COMMANDS cmd = d_commands[i];
        const float v = d_operands[i];

        switch (cmd) {
            case CALCULATOR_ADD: {
                out += v;
                break;
            }
            case CALCULATOR_SUB: {
                out -= v;
                break;
            }
            case CALCULATOR_DIV: {
                out /= v;
                break;
            }
            case CALCULATOR_MUL: {
                out *= v;
                break;
            }
        }
    }

    output[idx] = out;
}

int main(int argc, char **argv) {
    CALCULATOR_COMMANDS h_commands[MAX_COMMANDS];
    float h_operands[MAX_COMMANDS];

    //get calculator operators from file
    const int num_commands = readCommandsFromFile(h_commands, h_operands);

    printf("%d commands found in file\n", num_commands);

    //copy commands and operands to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_commands), h_commands, sizeof(CALCULATOR_COMMANDS) * MAX_COMMANDS);
    checkCUDAError("Commands copy to constant memory");
    hipMemcpyToSymbol(HIP_SYMBOL(d_operands), h_operands, sizeof(float) * MAX_COMMANDS);
    checkCUDAError("Commands copy to constant memory");

    //perform fully synchronous version
    cudaCalculatorDefaultStream(h_commands, h_operands, num_commands);

    //perform asynchronous version
    cudaCalculatorNStream1(h_commands, h_operands, num_commands);

    //perform asynchronous version
    cudaCalculatorNStream2(h_commands, h_operands, num_commands);
}

void cudaCalculatorDefaultStream(CALCULATOR_COMMANDS *commands, float *operands, int num_commands) {
    float *d_input, *d_output;
    float time;
    hipEvent_t start, stop;

    //init cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocate memory
    float *h_input = static_cast<float *>(malloc(sizeof(float) * SAMPLES));
    float *h_output = static_cast<float *>(malloc(sizeof(float) * SAMPLES));

    //allocate device memory
    hipMalloc((void **)&d_input, sizeof(float) * SAMPLES);
    hipMalloc((void **)&d_output, sizeof(float) * SAMPLES);
    checkCUDAError("CUDA Memory allocate: default stream");

    //init the host input
    initInput(h_input);

    //begin timing
    hipEventRecord(start);

    //1) Asynchronous host to device memory copy
    hipMemcpy(d_input, h_input, sizeof(float) * SAMPLES, hipMemcpyHostToDevice);
    checkCUDAError("CUDA Memory copy H2D: default stream");

    //2) Execute kernel
    parallelCalculator << <SAMPLES / TPB, TPB >> > (d_input, d_output, num_commands);
    checkCUDAError("CUDA Kernel: default stream");

    //3) Asynchronousdevice to host memory copy
    hipMemcpy(h_output, d_output, sizeof(float) * SAMPLES, hipMemcpyDeviceToHost);
    checkCUDAError("CUDA Memory copy D2H: default stream");

    //end timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //check for errors and print timing
    const int errors = checkResults(h_input, h_output, commands, operands, num_commands);
    printf("Synchronous V Completed in %f seconds with %d errors\n", time, errors);

    //cleanup
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
}

void cudaCalculatorNStream1(CALCULATOR_COMMANDS *commands, float *operands, int num_commands) {
    float *h_input, *h_output;
    float *d_input, *d_output;
    float time;
    hipEvent_t start, stop;
    int i;
    hipStream_t streams[NUM_STREAMS];

    //init cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocate memory
    hipHostMalloc((void **)&h_input, sizeof(float) * SAMPLES);
    hipHostMalloc((void **)&h_output, sizeof(float) * SAMPLES);

    //allocate device memory
    hipMalloc((void **)&d_input, sizeof(float) * SAMPLES);
    hipMalloc((void **)&d_output, sizeof(float) * SAMPLES);
    checkCUDAError("CUDA Memory allocate: default stream");

    //init streams
    for (i = 0; i < NUM_STREAMS; i++) {
        //create the stream
        hipStreamCreate(&streams[i]);
    }

    //init the host input
    initInput(h_input);

    //begin timing
    hipEventRecord(start);

    int batch_samples = SAMPLES / NUM_STREAMS;
    for (i = 0; i < NUM_STREAMS; i++) {
        const int offset = i * batch_samples;

        //1) Asynchronous host to device memory copy
        hipMemcpyAsync(d_input + offset, h_input + offset, sizeof(float) * batch_samples, hipMemcpyHostToDevice, streams[i]);
        checkCUDAError("CUDA Memory copy H2D: N streams");

        //2) Execute kernel
        parallelCalculator << <batch_samples / TPB, TPB, 0, streams[i] >> > (d_input + offset, d_output + offset, num_commands);
        checkCUDAError("CUDA Kernel: N streams");

        //3) Asynchronous device to host memory copy
        hipMemcpyAsync(h_output + offset, d_output + offset, sizeof(float) * batch_samples, hipMemcpyDeviceToHost, streams[i]);
        checkCUDAError("CUDA Memory copy D2H: N streams");
    }

    //end timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //check for errors and print timing
    const int errors = checkResults(h_input, h_output, commands, operands, num_commands);
    printf("Async V1 (%d streams) Completed in %f seconds with %d errors\n", NUM_STREAMS, time, errors);

    //cleanup
    //init streams
    for (i = 0; i < NUM_STREAMS; i++) {
        //create the stream
        hipStreamDestroy(streams[i]);
    }
    hipFree(d_input);
    hipFree(d_output);
    hipHostFree(h_input);
    hipHostFree(h_output);
}

void cudaCalculatorNStream2(CALCULATOR_COMMANDS *commands, float *operands, int num_commands) {
    float *h_input, *h_output;
    float *d_input, *d_output;
    float time;
    hipEvent_t start, stop;
    int i;
    hipStream_t streams[NUM_STREAMS];

    //init cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocate memory
    hipHostMalloc((void **)&h_input, sizeof(float) * SAMPLES);
    hipHostMalloc((void **)&h_output, sizeof(float) * SAMPLES);

    //allocate device memory
    hipMalloc((void **)&d_input, sizeof(float) * SAMPLES);
    hipMalloc((void **)&d_output, sizeof(float) * SAMPLES);
    checkCUDAError("CUDA Memory allocate: default stream");

    //init streams
    for (i = 0; i < NUM_STREAMS; i++) {
        //create the stream
        hipStreamCreate(&streams[i]);
    }

    //init the host input
    initInput(h_input);

    //begin timing
    hipEventRecord(start);

    int batch_samples = SAMPLES / NUM_STREAMS;

    for (i = 0; i < NUM_STREAMS; i++) {
        const int offset = i * batch_samples;
        //1) Asynchronous host to device memory copy
        hipMemcpyAsync(d_input + offset, h_input + offset, sizeof(float) * batch_samples, hipMemcpyHostToDevice, streams[i]);
        checkCUDAError("CUDA Memory copy H2D: N streams");
    }

    for (i = 0; i < NUM_STREAMS; i++) {
        const int offset = i * batch_samples;
        //2) Execute kernel
        parallelCalculator << <batch_samples / TPB, TPB, 0, streams[i] >> > (d_input + offset, d_output + offset, num_commands);
        checkCUDAError("CUDA Kernel: N streams");
    }

    for (i = 0; i < NUM_STREAMS; i++) {
        const int offset = i * batch_samples;
        //3) Asynchronous device to host memory copy
        hipMemcpyAsync(h_output + offset, d_output + offset, sizeof(float) * batch_samples, hipMemcpyDeviceToHost, streams[i]);
        checkCUDAError("CUDA Memory copy D2H: N streams");
    }

    //end timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //check for errors and print timing
    const int errors = checkResults(h_input, h_output, commands, operands, num_commands);
    printf("Async V2 (%d streams) Completed in %f seconds with %d errors\n", NUM_STREAMS, time, errors);

    //cleanup
    //init streams
    for (i = 0; i < NUM_STREAMS; i++) {
        //create the stream
        hipStreamDestroy(streams[i]);
    }
    hipFree(d_input);
    hipFree(d_output);
    hipHostFree(h_input);
    hipHostFree(h_output);
}

int readCommandsFromFile(CALCULATOR_COMMANDS *commands, float *operands) {
    float in_value;
    char buffer[FILE_BUFFER_SIZE];
    char command[4];
    unsigned int line = 0;

    printf("Welcome to the COM4521 Parallel floating point Calculator\n");
    FILE *f = fopen("commands.calc", "r");
    if (f == nullptr) {
        fprintf(stderr, "File not found\n");
        return 0;
    }

    while (readLine(f, buffer)) {
        line++;

        if (line >= MAX_COMMANDS) {
            fprintf(stderr, "To many commands in form maximum is %u\n", MAX_COMMANDS);
            return 0;
        }

        if (!(isalpha(buffer[0]) && isalpha(buffer[1]) && isalpha(buffer[2]) && buffer[3] == ' ')) {
            fprintf(stderr, "Incorrect command format at line %u\n", line);
            return 0;
        }

        sscanf(buffer, "%s %f", command, &in_value);

        if (strcmp(command, "add") == 0) {
            commands[line] = CALCULATOR_ADD;
        } else if (strcmp(command, "sub") == 0) {
            commands[line] = CALCULATOR_SUB;
        } else if (strcmp(command, "div") == 0) {
            commands[line] = CALCULATOR_DIV;
        } else if (strcmp(command, "mul") == 0) {
            commands[line] = CALCULATOR_MUL;
        } else {
            fprintf(stderr, "Unknown command at line %u!\n", line);
            return 0;
        }

        operands[line] = in_value;
    }

    fclose(f);

    return line;
}

void initInput(float *input) {
    for (int i = 0; i < SAMPLES; i++) {
        if (INPUT == INPUT_LINEAR) {
            input[i] = static_cast<float>(i);
        } else if (INPUT == INPUT_RANDOM) {
            input[i] = rand() / static_cast<float>(RAND_MAX);
        }
    }
}

int readLine(FILE *f, char buffer[]) {
    int i = 0;
    char c;
    while ((c = getc(f)) != '\n') {
        if (c == EOF)
            return 0;
        buffer[i++] = c;
        if (i == FILE_BUFFER_SIZE) {
            fprintf(stderr, "Buffer size is too small for line input\n");
            exit(0);
        }
    }
    buffer[i] = '\0';

    if (strncmp(buffer, "exit", 4) == 0) {
        return 0;
    }

    return 1;
}

int checkResults(const float *h_input, float *h_output, const CALCULATOR_COMMANDS *commands, const float *operands, int num_commands) {
    int errors = 0;

    for (int i = 0; i < SAMPLES; i++) {
        float out = h_input[i];

        for (int j = 0; j < num_commands; j++) {
            const CALCULATOR_COMMANDS cmd = commands[j];
            const float v = operands[j];

            switch (cmd) {
                case CALCULATOR_ADD: {
                    out += v;
                    break;
                }
                case CALCULATOR_SUB: {
                    out -= v;
                    break;
                }
                case CALCULATOR_DIV: {
                    out /= v;
                    break;
                }
                case CALCULATOR_MUL: {
                    out *= v;
                    break;
                }
            }
        }

        //test the result
        if (h_output[i] != out) {
            //fprintf(stderr, "Error: GPU result (%f) differs from CPU result (%f) at index %d\n", h_output[i], out, i);
            errors++;
        }
    }

    return errors;
}

void checkCUDAError(const char *msg) {
    const hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
